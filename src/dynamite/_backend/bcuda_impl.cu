#include "hip/hip_runtime.h"

#include "bcuda_impl.h"

#ifdef PETSC_USE_64BIT_INDICES
  #define CUDA_POPCOUNT(x) (__popcll(x))
  #define CUDA_CTZ(x) (__ffsll(x)-1)
#else
  #define CUDA_POPCOUNT(x) (__popc(x))
  #define CUDA_CTZ(x) (__ffs(x)-1)
#endif

#define CUDA_PARITY(x) (CUDA_POPCOUNT(x)&1)

#define TERM_REAL_CUDA(mask, sign) (!CUDA_PARITY((mask) & (sign)))

/* subspace functions for the GPU */

PetscErrorCode CopySubspaceData_CUDA_Full(data_Full** out_p, const data_Full* in) {
  PetscCallCUDA(hipMalloc((void **) out_p, sizeof(data_Full)));
  PetscCallCUDA(hipMemcpy(*out_p, in, sizeof(data_Full), hipMemcpyHostToDevice));
  return 0;
}

PetscErrorCode DestroySubspaceData_CUDA_Full(data_Full* data) {
  PetscCallCUDA(hipFree(data));
  return 0;
}

__device__ PetscInt S2I_CUDA_Full(PetscInt state, const data_Full* data) {
  return state;
}

__device__ PetscInt I2S_CUDA_Full(PetscInt idx, const data_Full* data) {
  return idx;
}

PetscErrorCode CopySubspaceData_CUDA_Parity(data_Parity** out_p, const data_Parity* in) {
  PetscCallCUDA(hipMalloc((void **) out_p, sizeof(data_Parity)));
  PetscCallCUDA(hipMemcpy(*out_p, in, sizeof(data_Parity), hipMemcpyHostToDevice));
  return 0;
}

PetscErrorCode DestroySubspaceData_CUDA_Parity(data_Parity* data) {
  PetscCallCUDA(hipFree(data));
  return 0;
}

__device__ PetscInt S2I_CUDA_Parity(PetscInt state, const data_Parity* data) {
  return (CUDA_PARITY(state) == data->space) ? state>>1 : (PetscInt)(-1);
}

__device__ PetscInt I2S_CUDA_Parity(PetscInt idx, const data_Parity* data) {
  return (idx<<1) | (CUDA_PARITY(idx) ^ data->space);
}

PetscErrorCode CopySubspaceData_CUDA_SpinConserve(data_SpinConserve** out_p, const data_SpinConserve* in) {
  PetscInt len_nchoosek = (in->k+1)*in->ld_nchoosek;

  data_SpinConserve cpu_data;

  PetscCall(PetscMemcpy(&cpu_data, in, sizeof(data_SpinConserve)));

  PetscCallCUDA(hipMalloc(&(cpu_data.nchoosek), sizeof(PetscInt)*len_nchoosek));
  PetscCallCUDA(hipMemcpy(cpu_data.nchoosek, in->nchoosek,
                           sizeof(PetscInt)*len_nchoosek, hipMemcpyHostToDevice));

  PetscCallCUDA(hipMalloc((void **) out_p, sizeof(data_SpinConserve)));
  PetscCallCUDA(hipMemcpy(*out_p, &cpu_data, sizeof(data_SpinConserve), hipMemcpyHostToDevice));

  return 0;
}

PetscErrorCode DestroySubspaceData_CUDA_SpinConserve(data_SpinConserve* data) {
  data_SpinConserve cpu_data;

  PetscCallCUDA(hipMemcpy(&cpu_data, data, sizeof(data_SpinConserve), hipMemcpyDeviceToHost));

  PetscCallCUDA(hipFree(cpu_data.nchoosek));
  PetscCallCUDA(hipFree(data));
  return 0;
}

__device__ PetscInt S2I_CUDA_SpinConserve(PetscInt state, const data_SpinConserve* data) {
  PetscInt n, k=0, idx=0;

  if (state >> data->L) return (PetscInt)(-1);
  if (CUDA_POPCOUNT(state) != data->k) return (PetscInt)(-1);

  while (state) {
    n = CUDA_CTZ(state);
    k++;
    if (k <= n) idx += data->nchoosek[k*data->ld_nchoosek + n];
    state &= state-1;  // pop least significant bit off of state
  }

  return idx;
}

__device__ PetscInt I2S_CUDA_SpinConserve(PetscInt idx, const data_SpinConserve* data) {
  PetscInt state = 0;
  PetscInt k = data->k;
  PetscInt current;
  for (PetscInt n=data->L; n>0; --n) {
    state <<= 1;
    current = (k > n-1) ? 0 : data->nchoosek[k*data->ld_nchoosek + n-1];
    if (idx >= current) {
        idx -= current;
        k--;
        state |= 1 ;
    }
  }
  return state;
}

PetscErrorCode CopySubspaceData_CUDA_Explicit(data_Explicit** out_p, const data_Explicit* in) {
  data_Explicit cpu_data;

  PetscCall(PetscMemcpy(&cpu_data, in, sizeof(data_Explicit)));

  PetscCallCUDA(hipMalloc(&(cpu_data.state_map), sizeof(PetscInt)*in->dim));
  PetscCallCUDA(hipMemcpy(cpu_data.state_map, in->state_map,
    sizeof(PetscInt)*in->dim, hipMemcpyHostToDevice));

  if (in->rmap_indices != PETSC_NULLPTR) {
    PetscCallCUDA(hipMalloc(&(cpu_data.rmap_indices), sizeof(PetscInt)*in->dim));
    PetscCallCUDA(hipMemcpy(cpu_data.rmap_indices, in->rmap_indices,
                             sizeof(PetscInt)*in->dim, hipMemcpyHostToDevice));
  }

  PetscCallCUDA(hipMalloc(&(cpu_data.rmap_states), sizeof(PetscInt)*in->dim));
  PetscCallCUDA(hipMemcpy(cpu_data.rmap_states, in->rmap_states,
    sizeof(PetscInt)*in->dim, hipMemcpyHostToDevice));

  PetscCallCUDA(hipMalloc((void **) out_p, sizeof(data_Explicit)));
  PetscCallCUDA(hipMemcpy(*out_p, &cpu_data, sizeof(data_Explicit), hipMemcpyHostToDevice));

  return 0;
}

PetscErrorCode DestroySubspaceData_CUDA_Explicit(data_Explicit* data) {
  data_Explicit cpu_data;

  PetscCallCUDA(hipMemcpy(&cpu_data, data, sizeof(data_Explicit), hipMemcpyDeviceToHost));

  PetscCallCUDA(hipFree(cpu_data.state_map));
  if (cpu_data.rmap_indices != PETSC_NULLPTR) {
    PetscCallCUDA(hipFree(cpu_data.rmap_indices));
  }
  PetscCallCUDA(hipFree(cpu_data.rmap_states));
  PetscCallCUDA(hipFree(data));
  return 0;
}

/* TODO: this is really not well suited for GPUs */
/* but I bet we can do something clever! */
__device__ PetscInt S2I_CUDA_Explicit(PetscInt state, const data_Explicit* data) {
  PetscInt left, right, mid;
  left = 0;
  right = data->dim;
  while (left <= right) {
    mid = left + (right-left)/2;
    if (data->rmap_states[mid] == state) {
      if (data->rmap_indices != PETSC_NULLPTR) {
        return data->rmap_indices[mid];
      } else {
        return mid;
      }
    }

    if (data->rmap_states[mid] < state) {
      left = mid + 1;
    }
    else {
      right = mid - 1;
    }
  }
  /* element was not in the array */
  return -1;
}

__device__ PetscInt I2S_CUDA_Explicit(PetscInt idx, const data_Explicit* data) {
  return data->state_map[idx];
}

PetscErrorCode MatCreateVecs_GPU(Mat mat, Vec *right, Vec *left)
{
  PetscInt M, N;

  PetscCall(MatGetSize(mat, &M, &N));

  if (right) {
    PetscCall(VecCreate(PetscObjectComm((PetscObject)mat),right));
    PetscCall(VecSetSizes(*right, PETSC_DECIDE, N));
    PetscCall(VecSetFromOptions(*right));
  }
  if (left) {
    PetscCall(VecCreate(PetscObjectComm((PetscObject)mat),left));
    PetscCall(VecSetSizes(*left, PETSC_DECIDE, M));
    PetscCall(VecSetFromOptions(*left));
  }

  return 0;
}

__device__ static __inline__ void add_real(PetscScalar *x, PetscReal r) {
  PetscReal *real_part;
  real_part = (PetscReal*) x;
  (*real_part) += r;
}

__device__ static __inline__ void add_imag(PetscScalar *x, PetscReal c) {
  PetscReal *imag_part;
  imag_part = ((PetscReal*)x) + 1;
  (*imag_part) += c;
}

// defines used in the various templates
#define Full_SP 0
#define Parity_SP 1
#define SpinConserve_SP 2
#define Explicit_SP 3

#define SUBSPACE Full
  #include "bcuda_template_1.cu"
#undef SUBSPACE

#define SUBSPACE Parity
  #include "bcuda_template_1.cu"
#undef SUBSPACE

#define SUBSPACE SpinConserve
  #include "bcuda_template_1.cu"
#undef SUBSPACE

#define SUBSPACE Explicit
  #include "bcuda_template_1.cu"
#undef SUBSPACE

#define LEFT_SUBSPACE Full
  #define RIGHT_SUBSPACE Full
    #include "bcuda_template_2.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE Parity
    #include "bcuda_template_2.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE SpinConserve
    #include "bcuda_template_2.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE Explicit
    #include "bcuda_template_2.cu"
  #undef RIGHT_SUBSPACE
#undef LEFT_SUBSPACE

#define LEFT_SUBSPACE Parity
  #define RIGHT_SUBSPACE Full
    #include "bcuda_template_2.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE Parity
    #include "bcuda_template_2.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE SpinConserve
    #include "bcuda_template_2.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE Explicit
    #include "bcuda_template_2.cu"
  #undef RIGHT_SUBSPACE
#undef LEFT_SUBSPACE

#define LEFT_SUBSPACE SpinConserve
  #define RIGHT_SUBSPACE Full
    #include "bcuda_template_2.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE Parity
    #include "bcuda_template_2.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE SpinConserve
    #include "bcuda_template_2.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE Explicit
    #include "bcuda_template_2.cu"
  #undef RIGHT_SUBSPACE
#undef LEFT_SUBSPACE

#define LEFT_SUBSPACE Explicit
  #define RIGHT_SUBSPACE Full
    #include "bcuda_template_2.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE Parity
    #include "bcuda_template_2.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE SpinConserve
    #include "bcuda_template_2.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE Explicit
    #include "bcuda_template_2.cu"
  #undef RIGHT_SUBSPACE
#undef LEFT_SUBSPACE
