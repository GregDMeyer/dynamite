#include "hip/hip_runtime.h"

#include "bcuda_impl.h"

#ifdef PETSC_USE_64BIT_INDICES
  #define CUDA_POPCOUNT(x) (__popcll(x))
  #define CUDA_CTZ(x) (__ffsll(x)-1)
#else
  #define CUDA_POPCOUNT(x) (__popc(x))
  #define CUDA_CTZ(x) (__ffs(x)-1)
#endif

#define CUDA_PARITY(x) (CUDA_POPCOUNT(x)&1)

#define TERM_REAL_CUDA(mask, sign) (!CUDA_PARITY((mask) & (sign)))

/* subspace functions for the GPU */

PetscErrorCode CopySubspaceData_CUDA_Full(data_Full** out_p, const data_Full* in) {
  hipError_t err;
  err = hipMalloc((void **) out_p, sizeof(data_Full));CHKERRCUDA(err);
  err = hipMemcpy(*out_p, in, sizeof(data_Full), hipMemcpyHostToDevice);CHKERRCUDA(err);
  return 0;
}

PetscErrorCode DestroySubspaceData_CUDA_Full(data_Full* data) {
  hipError_t err;
  err = hipFree(data);CHKERRCUDA(err);
  return 0;
}

__device__ PetscInt S2I_CUDA_Full(PetscInt state, const data_Full* data) {
  return state;
}

__device__ PetscInt I2S_CUDA_Full(PetscInt idx, const data_Full* data) {
  return idx;
}

PetscErrorCode CopySubspaceData_CUDA_Parity(data_Parity** out_p, const data_Parity* in) {
  hipError_t err;
  err = hipMalloc((void **) out_p, sizeof(data_Parity));CHKERRCUDA(err);
  err = hipMemcpy(*out_p, in, sizeof(data_Parity), hipMemcpyHostToDevice);CHKERRCUDA(err);
  return 0;
}

PetscErrorCode DestroySubspaceData_CUDA_Parity(data_Parity* data) {
  hipError_t err;
  err = hipFree(data);CHKERRCUDA(err);
  return 0;
}

__device__ PetscInt S2I_CUDA_Parity(PetscInt state, const data_Parity* data) {
  return (CUDA_PARITY(state) == data->space) ? state>>1 : (PetscInt)(-1);
}

__device__ PetscInt I2S_CUDA_Parity(PetscInt idx, const data_Parity* data) {
  return (idx<<1) | (CUDA_PARITY(idx) ^ data->space);
}

PetscErrorCode CopySubspaceData_CUDA_SpinConserve(data_SpinConserve** out_p, const data_SpinConserve* in) {
  hipError_t err;
  PetscInt len_nchoosek = (in->k+1)*in->ld_nchoosek;

  data_SpinConserve cpu_data;

  PetscCall(PetscMemcpy(&cpu_data, in, sizeof(data_SpinConserve)));

  err = hipMalloc(&(cpu_data.nchoosek), sizeof(PetscInt)*len_nchoosek);CHKERRCUDA(err);
  err = hipMemcpy(cpu_data.nchoosek, in->nchoosek,
		   sizeof(PetscInt)*len_nchoosek, hipMemcpyHostToDevice);CHKERRCUDA(err);

  err = hipMalloc((void **) out_p, sizeof(data_SpinConserve));CHKERRCUDA(err);
  err = hipMemcpy(*out_p, &cpu_data, sizeof(data_SpinConserve), hipMemcpyHostToDevice);CHKERRCUDA(err);

  return 0;
}

PetscErrorCode DestroySubspaceData_CUDA_SpinConserve(data_SpinConserve* data) {
  hipError_t err;

  data_SpinConserve cpu_data;

  err = hipMemcpy(&cpu_data, data, sizeof(data_SpinConserve), hipMemcpyDeviceToHost);CHKERRCUDA(err);

  err = hipFree(cpu_data.nchoosek);CHKERRCUDA(err);
  err = hipFree(data);CHKERRCUDA(err);
  return 0;
}

__device__ PetscInt S2I_CUDA_SpinConserve(PetscInt state, PetscInt* sign, const data_SpinConserve* data) {
  PetscInt n, k=0, idx=0;

  if (state >> data->L) return (PetscInt)(-1);
  if (CUDA_POPCOUNT(state) != data->k) return (PetscInt)(-1);

  while (state) {
    n = CUDA_CTZ(state);
    k++;
    if (k <= n) idx += data->nchoosek[k*data->ld_nchoosek + n];
    state &= state-1;  // pop least significant bit off of state
  }

  *sign = 1;
  PetscInt dim;
  if (data->spinflip) {
    dim = data->nchoosek[data->k*data->ld_nchoosek + data->L]/2;
    if (idx >= dim) {
      idx = 2*dim - idx - 1;
      *sign = data->spinflip;
    }
  }

  return idx;
}

__device__ PetscInt I2S_CUDA_SpinConserve(PetscInt idx, const data_SpinConserve* data) {
  PetscInt state = 0;
  PetscInt k = data->k;
  PetscInt current;
  for (PetscInt n=data->L; n>0; --n) {
    state <<= 1;
    current = (k > n-1) ? 0 : data->nchoosek[k*data->ld_nchoosek + n-1];
    if (idx >= current) {
        idx -= current;
        k--;
        state |= 1 ;
    }
  }
  return state;
}

PetscErrorCode CopySubspaceData_CUDA_Explicit(data_Explicit** out_p, const data_Explicit* in) {
  hipError_t err;

  data_Explicit cpu_data;

  PetscCall(PetscMemcpy(&cpu_data, in, sizeof(data_Explicit)));

  err = hipMalloc(&(cpu_data.state_map), sizeof(PetscInt)*in->dim);CHKERRCUDA(err);
  err = hipMemcpy(cpu_data.state_map, in->state_map,
    sizeof(PetscInt)*in->dim, hipMemcpyHostToDevice);CHKERRCUDA(err);

  err = hipMalloc(&(cpu_data.rmap_indices), sizeof(PetscInt)*in->dim);CHKERRCUDA(err);
  err = hipMemcpy(cpu_data.rmap_indices, in->rmap_indices,
    sizeof(PetscInt)*in->dim, hipMemcpyHostToDevice);CHKERRCUDA(err);

  err = hipMalloc(&(cpu_data.rmap_states), sizeof(PetscInt)*in->dim);CHKERRCUDA(err);
  err = hipMemcpy(cpu_data.rmap_states, in->rmap_states,
    sizeof(PetscInt)*in->dim, hipMemcpyHostToDevice);CHKERRCUDA(err);

  err = hipMalloc((void **) out_p, sizeof(data_Explicit));CHKERRCUDA(err);
  err = hipMemcpy(*out_p, &cpu_data, sizeof(data_Explicit), hipMemcpyHostToDevice);CHKERRCUDA(err);

  return 0;
}

PetscErrorCode DestroySubspaceData_CUDA_Explicit(data_Explicit* data) {
  hipError_t err;

  data_Explicit cpu_data;

  err = hipMemcpy(&cpu_data, data, sizeof(data_Explicit), hipMemcpyDeviceToHost);CHKERRCUDA(err);

  err = hipFree(cpu_data.state_map);CHKERRCUDA(err);
  err = hipFree(cpu_data.rmap_indices);CHKERRCUDA(err);
  err = hipFree(cpu_data.rmap_states);CHKERRCUDA(err);
  err = hipFree(data);CHKERRCUDA(err);
  return 0;
}

/* TODO: this is really not well suited for GPUs */
/* but I bet we can do something clever! */
__device__ PetscInt S2I_CUDA_Explicit(PetscInt state, const data_Explicit* data) {
  PetscInt left, right, mid;
  left = 0;
  right = data->dim;
  while (left <= right) {
    mid = left + (right-left)/2;
    if (data->rmap_states[mid] == state) {
      return data->rmap_indices[mid];
    }

    if (data->rmap_states[mid] < state) {
      left = mid + 1;
    }
    else {
      right = mid - 1;
    }
  }
  /* element was not in the array */
  return -1;
}

__device__ PetscInt I2S_CUDA_Explicit(PetscInt idx, const data_Explicit* data) {
  return data->state_map[idx];
}

PetscErrorCode MatCreateVecs_GPU(Mat mat, Vec *right, Vec *left)
{
  PetscInt M, N;

  PetscCall(MatGetSize(mat, &M, &N));

  if (right) {
    PetscCall(VecCreate(PetscObjectComm((PetscObject)mat),right));
    PetscCall(VecSetSizes(*right, PETSC_DECIDE, N));
    PetscCall(VecSetFromOptions(*right));
  }
  if (left) {
    PetscCall(VecCreate(PetscObjectComm((PetscObject)mat),left));
    PetscCall(VecSetSizes(*left, PETSC_DECIDE, M));
    PetscCall(VecSetFromOptions(*left));
  }

  return 0;
}

__device__ static __inline__ void add_real(PetscScalar *x, PetscReal r) {
  PetscReal *real_part;
  real_part = (PetscReal*) x;
  (*real_part) += r;
}

__device__ static __inline__ void add_imag(PetscScalar *x, PetscReal c) {
  PetscReal *imag_part;
  imag_part = ((PetscReal*)x) + 1;
  (*imag_part) += c;
}

// defines used in the various templates
#define Full_SP 0
#define Parity_SP 1
#define SpinConserve_SP 2
#define Explicit_SP 3

#define LEFT_SUBSPACE Full
  #define RIGHT_SUBSPACE Full
    #include "bcuda_template.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE Parity
    #include "bcuda_template.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE SpinConserve
    #include "bcuda_template.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE Explicit
    #include "bcuda_template.cu"
  #undef RIGHT_SUBSPACE
#undef LEFT_SUBSPACE

#define LEFT_SUBSPACE Parity
  #define RIGHT_SUBSPACE Full
    #include "bcuda_template.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE Parity
    #include "bcuda_template.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE SpinConserve
    #include "bcuda_template.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE Explicit
    #include "bcuda_template.cu"
  #undef RIGHT_SUBSPACE
#undef LEFT_SUBSPACE

#define LEFT_SUBSPACE SpinConserve
  #define RIGHT_SUBSPACE Full
    #include "bcuda_template.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE Parity
    #include "bcuda_template.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE SpinConserve
    #include "bcuda_template.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE Explicit
    #include "bcuda_template.cu"
  #undef RIGHT_SUBSPACE
#undef LEFT_SUBSPACE

#define LEFT_SUBSPACE Explicit
  #define RIGHT_SUBSPACE Full
    #include "bcuda_template.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE Parity
    #include "bcuda_template.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE SpinConserve
    #include "bcuda_template.cu"
  #undef RIGHT_SUBSPACE

  #define RIGHT_SUBSPACE Explicit
    #include "bcuda_template.cu"
  #undef RIGHT_SUBSPACE
#undef LEFT_SUBSPACE
