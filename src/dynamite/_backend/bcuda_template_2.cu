#include "hip/hip_runtime.h"

#include "bcuda_template_2_private.h"

PetscErrorCode C(BuildGPUShell,C(LEFT_SUBSPACE,RIGHT_SUBSPACE))(
  const msc_t *msc,
  const C(data,LEFT_SUBSPACE)* left_subspace_data,
  const C(data,RIGHT_SUBSPACE)* right_subspace_data,
  int xparity,
  Mat *A)
{
  PetscInt M, N, mpi_size;
  shell_context *ctx;

  PetscCallMPI(MPI_Comm_size(PETSC_COMM_WORLD, &mpi_size));
  if (mpi_size > 1) {
    SETERRQ(PETSC_COMM_WORLD, PETSC_ERR_SUP,
      "Shell GPU matrices currently only implemented for 1 MPI process.");
  }

  /* N is dimension of right subspace, M of left */
  M = C(Dim,LEFT_SUBSPACE)(left_subspace_data);
  N = C(Dim,RIGHT_SUBSPACE)(right_subspace_data);
  if (xparity) {
    M /= 2;
    N /= 2;
  }

  PetscCall(C(BuildContext_CUDA,C(LEFT_SUBSPACE,RIGHT_SUBSPACE))(
    msc, left_subspace_data, right_subspace_data, &ctx));

  PetscCall(MatCreateShell(PETSC_COMM_WORLD, M, N, M, N, ctx, A));

  PetscCall(MatShellSetOperation(*A, MATOP_MULT,
    (void(*)(void))C(MatMult_GPU,C(LEFT_SUBSPACE,RIGHT_SUBSPACE))));
  PetscCall(MatShellSetOperation(*A, MATOP_NORM,
    (void(*)(void))C(MatNorm_GPU,C(LEFT_SUBSPACE,RIGHT_SUBSPACE))));
  PetscCall(MatShellSetOperation(*A, MATOP_CREATE_VECS,
    (void(*)(void))MatCreateVecs_GPU));
  PetscCall(MatShellSetOperation(*A, MATOP_DESTROY,
    (void(*)(void))C(MatDestroyCtx_GPU,C(LEFT_SUBSPACE,RIGHT_SUBSPACE))));

  return 0;
}

PetscErrorCode C(BuildContext_CUDA,C(LEFT_SUBSPACE,RIGHT_SUBSPACE))(
  const msc_t *msc,
  const C(data,LEFT_SUBSPACE)* left_subspace_data,
  const C(data,RIGHT_SUBSPACE)* right_subspace_data,
  shell_context **ctx_p)
{
  /* NOTE: some data shared by GPU and CPU implementations is set in BuildMat */

  PetscReal *cpu_real_coeffs, real_part;
  PetscInt nterms, i;
  shell_context *ctx;

  PetscCall(PetscMalloc(sizeof(shell_context), ctx_p));
  ctx = (*ctx_p);

  ctx->gpu = PETSC_TRUE;
  nterms = msc->mask_offsets[msc->nmasks];

  PetscCallCUDA(hipMalloc((void **) &(ctx->masks),
    sizeof(PetscInt)*msc->nmasks));
  PetscCallCUDA(hipMemcpy(ctx->masks, msc->masks, sizeof(PetscInt)*msc->nmasks,
    hipMemcpyHostToDevice));

  PetscCallCUDA(hipMalloc((void **) &(ctx->mask_offsets),
    sizeof(PetscInt)*(msc->nmasks+1)));
  PetscCallCUDA(hipMemcpy(ctx->mask_offsets, msc->mask_offsets, sizeof(PetscInt)*(msc->nmasks+1),
    hipMemcpyHostToDevice));

  PetscCallCUDA(hipMalloc((void **) &(ctx->signs), sizeof(PetscInt)*nterms));
  PetscCallCUDA(hipMemcpy(ctx->signs, msc->signs, sizeof(PetscInt)*nterms,
    hipMemcpyHostToDevice));

  PetscCallCUDA(hipMalloc((void **) &(ctx->real_coeffs), sizeof(PetscReal)*nterms));
  /*
   * we need a CPU vector in which we will store the real coefficients, then we'll copy
   * from that over to the CPU.
   */
  PetscCall(PetscMalloc1(nterms, &cpu_real_coeffs));
  for (i=0; i < nterms; ++i) {
    real_part = PetscRealPart(msc->coeffs[i]);
    cpu_real_coeffs[i] = (real_part != 0) ? real_part : PetscImaginaryPart(msc->coeffs[i]);
  }
  PetscCallCUDA(hipMemcpy(ctx->real_coeffs, cpu_real_coeffs, sizeof(PetscReal)*nterms,
    hipMemcpyHostToDevice));
  PetscCall(PetscFree(cpu_real_coeffs));

  PetscCall(C(CopySubspaceData_CUDA,LEFT_SUBSPACE)(
    (C(data,LEFT_SUBSPACE)**)&(ctx->left_subspace_data),
    (C(data,LEFT_SUBSPACE)*)left_subspace_data));
  PetscCall(C(CopySubspaceData_CUDA,RIGHT_SUBSPACE)(
    (C(data,RIGHT_SUBSPACE)**)&(ctx->right_subspace_data),
    (C(data,RIGHT_SUBSPACE)*)right_subspace_data));

  return 0;
}

PetscErrorCode C(MatDestroyCtx_GPU,C(LEFT_SUBSPACE,RIGHT_SUBSPACE))(Mat A)
{
  shell_context *ctx;

  PetscCall(MatShellGetContext(A, &ctx));

  PetscCallCUDA(hipFree(ctx->masks));
  PetscCallCUDA(hipFree(ctx->mask_offsets));
  PetscCallCUDA(hipFree(ctx->signs));
  PetscCallCUDA(hipFree(ctx->real_coeffs));

  if (ctx->diag) {
    PetscCallCUDA(hipFree(ctx->diag));
  }

  PetscCall(C(DestroySubspaceData_CUDA,LEFT_SUBSPACE)(
    (C(data,LEFT_SUBSPACE)*) ctx->left_subspace_data));
  PetscCall(C(DestroySubspaceData_CUDA,RIGHT_SUBSPACE)(
    (C(data,RIGHT_SUBSPACE)*) ctx->right_subspace_data));

  PetscCall(PetscFree(ctx));

  return 0;
}

PetscErrorCode C(MatMult_GPU,C(LEFT_SUBSPACE,RIGHT_SUBSPACE))(Mat A, Vec x, Vec b)
{
  shell_context *ctx;

  const PetscScalar* xarray;
  PetscScalar* barray;
  PetscInt size;

  PetscCall(VecSet(b, 0));

  PetscCall(MatShellGetContext(A, &ctx));

  PetscCall(VecCUDAGetArrayRead(x, &xarray));
  PetscCall(VecCUDAGetArray(b, &barray));

  PetscCall(VecGetSize(b, &size));

  PetscCallCUDA(hipDeviceSynchronize());

  C(device_MatMult,C(LEFT_SUBSPACE,RIGHT_SUBSPACE))<<<GPU_BLOCK_NUM,GPU_BLOCK_SIZE>>>(
    size,
    ctx->masks,
    ctx->mask_offsets,
    ctx->signs,
    ctx->real_coeffs,
    ctx->nmasks,
    (C(data,LEFT_SUBSPACE)*) ctx->left_subspace_data,
    (C(data,RIGHT_SUBSPACE)*) ctx->right_subspace_data,
    ctx->diag,
    xarray,
    barray);

  PetscCallCUDA(hipDeviceSynchronize());

  PetscCall(VecCUDARestoreArrayRead(x, &xarray));
  PetscCall(VecCUDARestoreArray(b, &barray));

  return 0;
}

__global__ void C(device_MatMult,C(LEFT_SUBSPACE,RIGHT_SUBSPACE))(
  PetscInt size,
  PetscInt* masks,
  PetscInt* mask_offsets,
  PetscInt* signs,
  PetscReal* real_coeffs,
  PetscInt nmasks,
  C(data,LEFT_SUBSPACE) *left_subspace_data,
  C(data,RIGHT_SUBSPACE) *right_subspace_data,
  PetscReal* diag,
  const PetscScalar* xarray,
  PetscScalar* barray)
{

  /* the following four lines come from the PETSc cuda source */
  PetscInt entries_per_group = (size - 1) / gridDim.x + 1;
  entries_per_group = (entries_per_group == 0) ? 1 : entries_per_group;  // for very small vectors, a group should still do some work
  PetscInt vec_start_index = blockIdx.x * entries_per_group;
  PetscInt vec_stop_index  = PetscMin((blockIdx.x + 1) * entries_per_group, size); // don't go beyond vec size

  PetscScalar tmp, val;
  PetscReal sign;
  PetscInt bra, ket, row_idx, col_idx, mask_idx, term_idx, this_start;

  this_start = vec_start_index + threadIdx.x;

  for (row_idx = this_start; row_idx < vec_stop_index; row_idx += blockDim.x) {
    ket = C(I2S_CUDA,LEFT_SUBSPACE)(row_idx,left_subspace_data);

    if (diag) {
      val = diag[row_idx] * xarray[row_idx];
      mask_idx = 1;
    } else {
      val = 0;
      mask_idx = 0;
    }

    for (; mask_idx<nmasks; ++mask_idx) {
      tmp = 0;
      bra = ket ^ masks[mask_idx];

      col_idx = C(S2I_CUDA,RIGHT_SUBSPACE)(bra, right_subspace_data);
      if (col_idx == -1) {  // state is outside of the subspace; skip it
        continue;
      }

      /* sum all terms for this matrix element */
      for (term_idx = mask_offsets[mask_idx]; term_idx < mask_offsets[mask_idx+1]; ++term_idx) {
#if defined(PETSC_USE_64BIT_INDICES)
        sign = __popcll(bra & signs[term_idx])&1;
#else
        sign = __popc(bra & signs[term_idx])&1;
#endif
        sign = 1 - 2*sign;
        if TERM_REAL_CUDA(masks[mask_idx], signs[term_idx]) {
	  add_real(&tmp, sign * real_coeffs[term_idx]);
        }
        else {
          add_imag(&tmp, sign * real_coeffs[term_idx]);
        }
      }
      val += tmp * xarray[col_idx];
    }

    barray[row_idx] = val;

  }
}

PetscErrorCode C(MatNorm_GPU,C(LEFT_SUBSPACE,RIGHT_SUBSPACE))(Mat A, NormType type, PetscReal *nrm)
{
  shell_context *ctx;

  PetscReal *d_maxs,*h_maxs;
  PetscInt i, M;

  if (type != NORM_INFINITY) {
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_ARG_OUTOFRANGE,"Only NORM_INFINITY is implemented for shell matrices.");
  }

  PetscCall(MatShellGetContext(A, &ctx));

  /*
    keep the norm cached so we don't have to compute it all the time.
    if we already have it, just return it
  */
  if (ctx->nrm != -1) {
    (*nrm) = ctx->nrm;
    return 0;
  }

  PetscCallCUDA(hipMalloc((void **) &d_maxs, sizeof(PetscReal)*GPU_BLOCK_NUM));
  PetscCall(PetscMalloc1(GPU_BLOCK_NUM, &h_maxs));

  PetscCall(MatGetSize(A, &M, NULL));

  C(device_MatNorm,C(LEFT_SUBSPACE,RIGHT_SUBSPACE))<<<GPU_BLOCK_NUM, GPU_BLOCK_SIZE, sizeof(PetscReal)*GPU_BLOCK_SIZE>>>(
    M,
    ctx->masks,
    ctx->mask_offsets,
    ctx->signs,
    ctx->real_coeffs,
    ctx->nmasks,
    (C(data,LEFT_SUBSPACE)*) ctx->left_subspace_data,
    (C(data,RIGHT_SUBSPACE)*) ctx->right_subspace_data,
    d_maxs);

  PetscCallCUDA(hipDeviceSynchronize());

  PetscCallCUDA(hipMemcpy(h_maxs, d_maxs, sizeof(PetscReal)*GPU_BLOCK_NUM, hipMemcpyDeviceToHost));

  /* now do max of h_maxs */
  (*nrm) = 0;
  for (i = 0; i < GPU_BLOCK_NUM; ++i) {
    if (h_maxs[i] > (*nrm)) (*nrm) = h_maxs[i];
  }

  ctx->nrm = (*nrm);

  PetscCallCUDA(hipFree(d_maxs));
  PetscCall(PetscFree(h_maxs));

  return 0;
}

__global__ void C(device_MatNorm,C(LEFT_SUBSPACE,RIGHT_SUBSPACE))(
  PetscInt size,
  PetscInt* masks,
  PetscInt* mask_offsets,
  PetscInt* signs,
  PetscReal* real_coeffs,
  PetscInt nmasks,
  C(data,LEFT_SUBSPACE) *left_subspace_data,
  C(data,RIGHT_SUBSPACE) *right_subspace_data,
  PetscReal *d_maxs)
{
  extern __shared__ PetscReal threadmax[];

  /* the following four lines come from the PETSc cuda source */
  PetscInt entries_per_group = (size - 1) / gridDim.x + 1;
  entries_per_group = (entries_per_group == 0) ? 1 : entries_per_group;  // for very small vectors, a group should still do some work
  PetscInt vec_start_index = blockIdx.x * entries_per_group;
  PetscInt vec_stop_index  = PetscMin((blockIdx.x + 1) * entries_per_group, size); // don't go beyond vec size

  PetscReal sum,v1,v2,sign;
  PetscScalar csum;
  PetscInt ket, bra, row_idx, mask_idx, term_idx, i;

  /* first find this thread's max and put it in threadmax */

  threadmax[threadIdx.x] = 0;
  for (row_idx = vec_start_index+threadIdx.x; row_idx < vec_stop_index; row_idx += blockDim.x) {
    ket = C(I2S_CUDA,LEFT_SUBSPACE)(row_idx,left_subspace_data);
    sum = 0;
    for (mask_idx = 0; mask_idx < nmasks; ++mask_idx) {
      csum = 0;
      bra = ket ^ masks[mask_idx];

      if (C(S2I_CUDA,RIGHT_SUBSPACE)(bra, right_subspace_data) == -1) {
	continue;
      }

      /* sum all terms for this matrix element */
      for (term_idx = mask_offsets[mask_idx]; term_idx < mask_offsets[mask_idx+1]; ++term_idx) {
#if defined(PETSC_USE_64BIT_INDICES)
        sign = __popcll(bra & signs[term_idx])&1;
#else
        sign = __popc(bra & signs[term_idx])&1;
#endif
        sign = 1 - 2*sign;
        if TERM_REAL_CUDA(masks[mask_idx], signs[term_idx]) {
	  add_real(&csum, sign * real_coeffs[term_idx]);
        }
        else {
          add_imag(&csum, sign * real_coeffs[term_idx]);
        }
      }
      sum += abs(csum);
    }
    if (sum > threadmax[threadIdx.x]) {
      threadmax[threadIdx.x] = sum;
    }
  }
  __syncthreads();

  /* now do the coolest reduce ever on the shared memory and hand it off to CPU */

  for (i=1; i<blockDim.x; i*=2) {
    if (threadIdx.x % (2*i) == 0) {
      v1 = threadmax[threadIdx.x];
      v2 = threadmax[threadIdx.x + i];
      threadmax[threadIdx.x] = v1>v2 ? v1 : v2;
    }
    __syncthreads();
  }

  if (threadIdx.x == 0) d_maxs[blockIdx.x] = threadmax[0];
}
