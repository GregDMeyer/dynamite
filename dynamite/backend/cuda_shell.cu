#include "hip/hip_runtime.h"

#include "cuda_shell_private.h"

extern "C" {
#include "cuda_shell.h"
}

PetscErrorCode BuildMat_CUDAShell(PetscInt L,PetscInt nterms,PetscInt* masks,PetscInt* signs,PetscScalar* coeffs,Mat *A)
{
  PetscErrorCode ierr;
  PetscInt N,n;
  shell_context *ctx;

  N = 1<<L;

  n = PETSC_DECIDE;
  PetscSplitOwnership(PETSC_COMM_WORLD,&n,&N);

  ierr = BuildContext_CUDA(L,nterms,masks,signs,coeffs,&ctx);CHKERRQ(ierr);

  ierr = MatCreateShell(PETSC_COMM_WORLD,n,n,N,N,ctx,A);CHKERRQ(ierr);
  ierr = MatShellSetOperation(*A,MATOP_MULT,(void(*)(void))MatMult_CUDAShell);
  ierr = MatShellSetOperation(*A,MATOP_NORM,(void(*)(void))MatNorm_CUDAShell);
  ierr = MatShellSetOperation(*A,MATOP_CREATE_VECS,(void(*)(void))MatCreateVecs_CUDAShell);

  return ierr;
}

PetscErrorCode BuildContext_CUDA(PetscInt L,PetscInt nterms,PetscInt* masks,PetscInt* signs,PetscScalar* coeffs,shell_context **ctx_p)
{
  PetscErrorCode ierr;
  hipError_t err;
  shell_context *ctx;

  ierr = PetscMalloc(sizeof(shell_context),ctx_p);CHKERRQ(ierr);
  ctx = (*ctx_p);

  ctx->L = L;
  ctx->nterms = nterms;
  ctx->nrm = -1;
  ctx->gpu = PETSC_TRUE;

  err = hipMalloc((void **) &(ctx->masks),  sizeof(PetscInt)*nterms);CHKERRCUDA(err);
  err = hipMalloc((void **) &(ctx->signs),  sizeof(PetscInt)*nterms);CHKERRCUDA(err);
  err = hipMalloc((void **) &(ctx->coeffs), sizeof(PetscScalar)*nterms);CHKERRCUDA(err);

  err = hipMemcpy(ctx->masks,masks,sizeof(PetscInt)*nterms,hipMemcpyHostToDevice);CHKERRCUDA(err);
  err = hipMemcpy(ctx->signs,signs,sizeof(PetscInt)*nterms,hipMemcpyHostToDevice);CHKERRCUDA(err);
  err = hipMemcpy(ctx->coeffs,coeffs,sizeof(PetscScalar)*nterms,hipMemcpyHostToDevice);CHKERRCUDA(err);

  return ierr;
}

PetscErrorCode DestroyContext_CUDA(Mat A)
{
  PetscErrorCode ierr;
  hipError_t err;
  shell_context *ctx;

  ierr = MatShellGetContext(A,&ctx);CHKERRQ(ierr);

  err = hipFree(ctx->masks);CHKERRCUDA(err);
  err = hipFree(ctx->signs);CHKERRCUDA(err);
  err = hipFree(ctx->coeffs);CHKERRCUDA(err);

  ierr = PetscFree(ctx);CHKERRQ(ierr);

  return ierr;
}

PetscErrorCode MatMult_CUDAShell(Mat M,Vec x,Vec b)
{
  PetscErrorCode ierr;
  hipError_t err;
  shell_context *ctx;

  const PetscScalar* xarray;
  PetscScalar* barray;
  PetscInt size;

  ierr = VecSet(b,0);CHKERRQ(ierr);

  ierr = MatShellGetContext(M,&ctx);CHKERRQ(ierr);

  ierr = VecCUDAGetArrayRead(x,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayReadWrite(b,&barray);CHKERRQ(ierr);

  size = 1 << ctx->L;

  err = hipDeviceSynchronize();CHKERRCUDA(err);

  device_MatMult_Shell<<<GPU_BLOCK_NUM,GPU_BLOCK_SIZE>>>(size,
                                                         ctx->masks,
                                                         ctx->signs,
                                                         ctx->coeffs,
                                                         ctx->nterms,
                                                         xarray,
                                                         barray);

  err = hipDeviceSynchronize();CHKERRCUDA(err);

  ierr = VecCUDARestoreArrayRead(x,&xarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayReadWrite(b,&barray);CHKERRQ(ierr);

  return ierr;
}

__global__ void device_MatMult_Shell(PetscInt size,
                                     PetscInt* masks,
                                     PetscInt* signs,
                                     PetscScalar* coeffs,
                                     PetscInt nterms,
                                     const PetscScalar* xarray,
                                     PetscScalar* barray)
{

  /* the following four lines come from the PETSc cuda source */
  PetscInt entries_per_group = (size - 1) / gridDim.x + 1;
  entries_per_group = (entries_per_group == 0) ? 1 : entries_per_group;  // for very small vectors, a group should still do some work
  PetscInt vec_start_index = blockIdx.x * entries_per_group;
  PetscInt vec_stop_index  = PetscMin((blockIdx.x + 1) * entries_per_group, size); // don't go beyond vec size

  PetscScalar tmp,val;
  PetscInt state,ket,mask,next_mask,this_start,i;

  this_start = vec_start_index + threadIdx.x;

  /* only access mask from global memory once */

  /* on the gpu, unlike on parallel CPUs, we have access
   * to the whole vector from any processor. That's awesome
   * because it means that we can accumulate results by row
   * instead of by column, and only do a single memory write
   * per entry in the output vector. Then we don't have to worry
   * about atomic operations either!
   */

  for (ket=this_start; ket<vec_stop_index; ket += blockDim.x) {
    val = 0;
    mask = masks[0];
    for (i=0;i<nterms;) {
      tmp = 0;
      state = ket ^ mask;
      /* sum all terms for this matrix element */
      do {
#if defined(PETSC_USE_64BIT_INDICES)
        tmp += __popcll(state & signs[i])%2 ? -coeffs[i] : coeffs[i];
#else
        tmp += __popc(state & signs[i])%2 ? -coeffs[i] : coeffs[i];
#endif
        ++i;
        if (i == nterms) break;
        next_mask = masks[i];
      } while (mask == next_mask);
      /* this can be optimized by keeping track of # of terms per matrix element.
         I think that should actually make it a lot faster because it gets rid of
         a significant chunk of the memory reads */
      val += tmp * xarray[state];
      mask = next_mask;
    }
    barray[ket] = val;
  }
}

PetscErrorCode MatNorm_CUDAShell(Mat A,NormType type,PetscReal *nrm)
{
  PetscErrorCode ierr;
  hipError_t err;
  shell_context *ctx;

  PetscReal *d_maxs,*h_maxs;
  PetscInt i,N;

  if (type != NORM_INFINITY) {
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_ARG_OUTOFRANGE,"Only NORM_INFINITY is implemented for shell matrices.");
  }

  ierr = MatShellGetContext(A,&ctx);CHKERRQ(ierr);

  /*
    keep the norm cached so we don't have to compute it all the time.
    if we already have it, just return it
  */
  if (ctx->nrm != -1) {
    (*nrm) = ctx->nrm;
    return ierr;
  }

  err = hipMalloc((void **) &d_maxs,sizeof(PetscReal)*GPU_BLOCK_NUM);CHKERRCUDA(err);
  ierr = PetscMalloc(sizeof(PetscReal)*GPU_BLOCK_NUM,&h_maxs);CHKERRQ(ierr);

  N = 1<<ctx->L;

  device_MatNorm_Shell<<<GPU_BLOCK_NUM,GPU_BLOCK_SIZE,sizeof(PetscReal)*GPU_BLOCK_SIZE>>>(N,ctx->masks,ctx->signs,ctx->coeffs,ctx->nterms,d_maxs);

  err = hipDeviceSynchronize();CHKERRCUDA(err);

  err = hipMemcpy(h_maxs,d_maxs,sizeof(PetscReal)*GPU_BLOCK_NUM,hipMemcpyDeviceToHost);CHKERRCUDA(err);

  /* now do max of h_maxs */
  (*nrm) = 0;
  for (i=0;i<GPU_BLOCK_NUM;++i) {
    if (h_maxs[i] > (*nrm)) (*nrm) = h_maxs[i];
  }

  ctx->nrm = (*nrm);

  err = hipFree(d_maxs);CHKERRCUDA(err);
  ierr = PetscFree(h_maxs);CHKERRQ(ierr);

  return ierr;
}

__global__ void device_MatNorm_Shell(PetscInt size,
                                     PetscInt* masks,
                                     PetscInt* signs,
                                     PetscScalar* coeffs,
                                     PetscInt nterms,
                                     PetscReal *d_maxs)
{
  extern __shared__ PetscReal threadmax[];

  /* the following four lines come from the PETSc cuda source */
  PetscInt entries_per_group = (size - 1) / gridDim.x + 1;
  entries_per_group = (entries_per_group == 0) ? 1 : entries_per_group;  // for very small vectors, a group should still do some work
  PetscInt vec_start_index = blockIdx.x * entries_per_group;
  PetscInt vec_stop_index  = PetscMin((blockIdx.x + 1) * entries_per_group, size); // don't go beyond vec size

  PetscReal sum,v1,v2;
  PetscScalar csum;
  PetscInt state, i, mask, next_mask;

  /* first find this thread's max and put it in threadmax */

  threadmax[threadIdx.x] = 0;
  for (state=vec_start_index+threadIdx.x;state<vec_stop_index;state += blockDim.x) {
    sum = 0;
    for (i=0;i<nterms;) {
      csum = 0;
      mask = masks[i];
      /* sum all terms for this matrix element */
      do {
#if defined(PETSC_USE_64BIT_INDICES)
        csum += __popcll(state & signs[i])%2 ? -coeffs[i] : coeffs[i];
#else
        csum += __popc(state & signs[i])%2 ? -coeffs[i] : coeffs[i];
#endif
        ++i;
        if (i >= nterms) break;
        next_mask = masks[i];
      } while (mask == next_mask);

      sum += abs(csum);
    }
    if (sum > threadmax[threadIdx.x]) {
      threadmax[threadIdx.x] = sum;
    }
  }
  __syncthreads();

  /* now do the coolest reduce ever on the shared memory and hand it off to CPU */

  for (i=1; i<blockDim.x; i*=2) {
    if (threadIdx.x % (2*i) == 0) {
      v1 = threadmax[threadIdx.x];
      v2 = threadmax[threadIdx.x + i];
      threadmax[threadIdx.x] = v1>v2 ? v1 : v2;
    }
    __syncthreads();
  }

  if (threadIdx.x == 0) d_maxs[blockIdx.x] = threadmax[0];
}

PetscErrorCode MatCreateVecs_CUDAShell(Mat mat, Vec *right, Vec *left)
{
  PetscErrorCode ierr;
  PetscInt N;

  ierr = MatGetSize(mat,&N,NULL);CHKERRQ(ierr);

  if (right) {
    ierr = VecCreate(PetscObjectComm((PetscObject)mat),right);CHKERRQ(ierr);
    ierr = VecSetSizes(*right,PETSC_DECIDE,N);CHKERRQ(ierr);
    ierr = VecSetFromOptions(*right);
  }
  if (left) {
    ierr = VecCreate(PetscObjectComm((PetscObject)mat),left);CHKERRQ(ierr);
    ierr = VecSetSizes(*left,PETSC_DECIDE,N);CHKERRQ(ierr);
    ierr = VecSetFromOptions(*left);
  }

  return 0;
}
